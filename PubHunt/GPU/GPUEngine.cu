#include "hip/hip_runtime.h"
/*
 * This file is part of the VanitySearch distribution (https://github.com/JeanLucPons/VanitySearch).
 * Copyright (c) 2019 Jean Luc PONS.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
*/

#include "GPUEngine.h"
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdint.h>
#include "../Timer.h"
#include "GPUMath.h"
#include "GPUHash.h"
#include "GPUCompute.h"
#include <string> // For std::string
#include <vector> // For std::vector in helpers
#include <stdexcept> // For std::runtime_error
#include <iomanip> // For std::setw, std::setfill
#include <sstream> // For std::ostringstream

// Forward declarations for functions used before they're defined
__host__ bool HostBN_HexToU64Array(const std::string& hex, uint64_t arr[4]);
__host__ uint64_t HostBN_Sub(uint64_t r[4], const uint64_t a[4], const uint64_t b[4]);
__host__ uint64_t HostBN_AddOneInplace(uint64_t r[4]);
__global__ void init_curand_states_kernel(hiprandStatePhilox4_32_10_t *states, unsigned long long seed, int num_states);
__global__ void generate_keys_in_range_kernel(uint64_t* output_keys, hiprandStatePhilox4_32_10_t* states, const uint64_t* dev_start_key, const uint64_t* dev_range_span, int num_keys_to_generate);

// ---------------------------------------------------------------------------------------

static const char* __cudaRandGetErrorEnum(hiprandStatus_t error) {
	switch (error) {
	case HIPRAND_STATUS_SUCCESS:
		return "HIPRAND_STATUS_SUCCESS";

	case HIPRAND_STATUS_VERSION_MISMATCH:
		return "HIPRAND_STATUS_VERSION_MISMATCH";

	case HIPRAND_STATUS_NOT_INITIALIZED:
		return "HIPRAND_STATUS_NOT_INITIALIZED";

	case HIPRAND_STATUS_ALLOCATION_FAILED:
		return "HIPRAND_STATUS_ALLOCATION_FAILED";

	case HIPRAND_STATUS_TYPE_ERROR:
		return "HIPRAND_STATUS_TYPE_ERROR";

	case HIPRAND_STATUS_OUT_OF_RANGE:
		return "HIPRAND_STATUS_OUT_OF_RANGE";

	case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
		return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";

	case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
		return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";

	case HIPRAND_STATUS_LAUNCH_FAILURE:
		return "HIPRAND_STATUS_LAUNCH_FAILURE";

	case HIPRAND_STATUS_PREEXISTING_FAILURE:
		return "HIPRAND_STATUS_PREEXISTING_FAILURE";

	case HIPRAND_STATUS_INITIALIZATION_FAILED:
		return "HIPRAND_STATUS_INITIALIZATION_FAILED";

	case HIPRAND_STATUS_ARCH_MISMATCH:
		return "HIPRAND_STATUS_ARCH_MISMATCH";

	case HIPRAND_STATUS_INTERNAL_ERROR:
		return "HIPRAND_STATUS_INTERNAL_ERROR";
	}

	return "<unknown>";
}

inline void __cudaRandSafeCall(hiprandStatus_t err, const char* file, const int line)
{
	if (HIPRAND_STATUS_SUCCESS != err)
	{
		fprintf(stderr, "CudaRandSafeCall() failed at %s:%i : %s\n", file, line, __cudaRandGetErrorEnum(err));
		exit(-1);
	}
	return;
}

inline void __cudaSafeCall(hipError_t err, const char* file, const int line)
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line, hipGetErrorString(err));
		exit(-1);
	}
	return;
}

#define CudaRandSafeCall( err ) __cudaRandSafeCall( err, __FILE__, __LINE__ )
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

// ---------------------------------------------------------------------------------------

__global__ void compute_hash(uint64_t* keys, uint32_t* hash160, int numHash160, uint32_t maxFound, uint32_t* found)
{

	int id = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
	ComputeHash(keys + id, hash160, numHash160, maxFound, found);

}

// ---------------------------------------------------------------------------------------

using namespace std;

int _ConvertSMVer2Cores(int major, int minor)
{

	// Defines for GPU Architecture types (using the SM version to determine
	// the # of cores per SM
	typedef struct {
		int SM;  // 0xMm (hexidecimal notation), M = SM Major version,
		// and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] = {
		{0x20, 32}, // Fermi Generation (SM 2.0) GF100 class
		{0x21, 48}, // Fermi Generation (SM 2.1) GF10x class
		{0x30, 192},
		{0x32, 192},
		{0x35, 192},
		{0x37, 192},
		{0x50, 128},
		{0x52, 128},
		{0x53, 128},
		{0x60,  64},
		{0x61, 128},
		{0x62, 128},
		{0x70,  64},
		{0x72,  64},
		{0x75,  64},
		{0x80,  64},
		{0x86, 128},
		{-1, -1}
	};

	int index = 0;

	while (nGpuArchCoresPerSM[index].SM != -1) {
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
			return nGpuArchCoresPerSM[index].Cores;
		}

		index++;
	}

	return 0;

}

// ----------------------------------------------------------------------------

GPUEngine::GPUEngine(int nbThreadGroup, int nbThreadPerGroup, int gpuId, uint32_t maxFound,
	const uint32_t* hash160, int numHash160,
	const std::string& startKeyHex, // Added
	const std::string& endKeyHex)   // Added
{
	this->dev_rand_states_ = nullptr;
	this->use_range_ = false;

	// Initialise CUDA
	this->nbThreadPerGroup = nbThreadPerGroup;
	this->numHash160 = numHash160;

	initialised = false;

	int deviceCount = 0;
	CudaSafeCall(hipGetDeviceCount(&deviceCount));

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0) {
		printf("GPUEngine: There are no available device(s) that support CUDA\n");
		exit(-1);
	}

	CudaSafeCall(hipSetDevice(gpuId));

	hipDeviceProp_t deviceProp;
	CudaSafeCall(hipGetDeviceProperties(&deviceProp, gpuId));

	if (nbThreadGroup == -1)
		nbThreadGroup = deviceProp.multiProcessorCount * 8;

	this->nbThread = nbThreadGroup * nbThreadPerGroup;
	this->maxFound = maxFound;
	this->outputSize = (maxFound * ITEM_SIZE_A + 4);

	char tmp[512];
	sprintf(tmp, "GPU #%d %s (%dx%d cores) Grid(%dx%d)",
		gpuId, deviceProp.name, deviceProp.multiProcessorCount,
		_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
		nbThread / nbThreadPerGroup,
		nbThreadPerGroup);
	deviceName = std::string(tmp);

	// Prefer L1 (We do not use __shared__ at all)
	CudaSafeCall(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

	size_t stackSize = 49152;
	CudaSafeCall(hipDeviceSetLimit(hipLimitStackSize, stackSize));

	// Allocate memory
	CudaSafeCall(hipMalloc((void**)&inputKey, nbThread * 4 * sizeof(uint64_t)));

	CudaSafeCall(hipMalloc((void**)&outputBuffer, outputSize));
	CudaSafeCall(hipHostAlloc(&outputBufferPinned, outputSize, hipHostMallocWriteCombined | hipHostMallocMapped));

	int K_SIZE = 5;

	CudaSafeCall(hipMalloc((void**)&inputHash, numHash160 * K_SIZE * sizeof(uint32_t)));
	CudaSafeCall(hipHostAlloc(&inputHashPinned, numHash160 * K_SIZE * sizeof(uint32_t), hipHostMallocWriteCombined | hipHostMallocMapped));

	memcpy(inputHashPinned, hash160, numHash160 * K_SIZE * sizeof(uint32_t));

	CudaSafeCall(hipMemcpy(inputHash, inputHashPinned, numHash160 * K_SIZE * sizeof(uint32_t), hipMemcpyHostToDevice));
	CudaSafeCall(hipHostFree(inputHashPinned));
	inputHashPinned = NULL;

	// Create a stream for non-blocking operations
	CudaSafeCall(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	
	// Skip cuRAND initialization which seems to be causing hangs
	// Just use simple randomize
	
	// Store key range if provided
	if (!startKeyHex.empty() && !endKeyHex.empty()) {
		uint64_t host_start_key[4], host_end_key[4], host_range_span[4];
		if (HostBN_HexToU64Array(startKeyHex, host_start_key) && 
			HostBN_HexToU64Array(endKeyHex, host_end_key)) {
			
			if (HostBN_Sub(host_range_span, host_end_key, host_start_key) == 1) { // if end < start (borrow occurred)
				printf("GPUEngine Error: End key must be greater than or equal to start key.\n");
				// For now, proceed without range.
				this->use_range_ = false;
			} else {
				HostBN_AddOneInplace(host_range_span); // range_span = end - start + 1

				CudaSafeCall(hipMalloc((void**)&dev_start_key_, 4 * sizeof(uint64_t)));
				CudaSafeCall(hipMemcpy(dev_start_key_, host_start_key, 4 * sizeof(uint64_t), hipMemcpyHostToDevice));

				CudaSafeCall(hipMalloc((void**)&dev_range_span_, 4 * sizeof(uint64_t)));
				CudaSafeCall(hipMemcpy(dev_range_span_, host_range_span, 4 * sizeof(uint64_t), hipMemcpyHostToDevice));
				
				this->use_range_ = true;
				printf("GPUEngine: Using key range %s to %s\n", startKeyHex.c_str(), endKeyHex.c_str());
			}
		} else {
			printf("GPUEngine Warning: Invalid hex string for range.Proceeding without range.\n");
			this->use_range_ = false;
		}
	}

	// Initialize with simple random data instead of using cuRAND
	Randomize();

	CudaSafeCall(hipGetLastError());
	initialised = true;
}

// ----------------------------------------------------------------------------

int GPUEngine::GetGroupSize()
{
	return GRP_SIZE;
}

// ----------------------------------------------------------------------------

void GPUEngine::PrintCudaInfo()
{
	const char* sComputeMode[] = {
		"Multiple host threads",
		"Only one host thread",
		"No host thread",
		"Multiple process threads",
		"Unknown",
		NULL
	};

	int deviceCount = 0;
	CudaSafeCall(hipGetDeviceCount(&deviceCount));

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0) {
		printf("GPUEngine: There are no available device(s) that support CUDA\n");
		return;
	}

	for (int i = 0; i < deviceCount; i++) {
		CudaSafeCall(hipSetDevice(i));
		hipDeviceProp_t deviceProp;
		CudaSafeCall(hipGetDeviceProperties(&deviceProp, i));
		printf("GPU #%d %s (%dx%d cores) (Cap %d.%d) (%.1f MB) (%s)\n",
			i, deviceProp.name, deviceProp.multiProcessorCount,
			_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
			deviceProp.major, deviceProp.minor, (double)deviceProp.totalGlobalMem / 1048576.0,
			sComputeMode[deviceProp.computeMode]);
	}
}

// ----------------------------------------------------------------------------

GPUEngine::~GPUEngine()
{
	CudaSafeCall(hipFree(inputKey));
	CudaSafeCall(hipFree(inputHash));

	CudaSafeCall(hipHostFree(outputBufferPinned));
	CudaSafeCall(hipFree(outputBuffer));

	CudaSafeCall(hipStreamDestroy(stream));

	if (use_range_) {
		CudaSafeCall(hipFree(dev_start_key_));
		CudaSafeCall(hipFree(dev_range_span_));
	}
	
	// Free cuRAND states if allocated
	if (dev_rand_states_ != nullptr) {
		CudaSafeCall(hipFree(dev_rand_states_));
		dev_rand_states_ = nullptr;
	}
}

// ----------------------------------------------------------------------------

int GPUEngine::GetNbThread()
{
	return nbThread;
}

// ----------------------------------------------------------------------------

bool GPUEngine::CallKernel()
{

	// Reset nbFound
	CudaSafeCall(hipMemset(outputBuffer, 0, 4));

	// Call the kernel (Perform STEP_SIZE keys per thread) 
	compute_hash << < nbThread / nbThreadPerGroup, nbThreadPerGroup >> >
		(inputKey, inputHash, numHash160, maxFound, outputBuffer);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("GPUEngine: callKernel: %s\n", hipGetErrorString(err));
		return false;
	}
	return true;

}

// ----------------------------------------------------------------------------

bool GPUEngine::Step(std::vector<ITEM>& dataFound, bool spinWait)
{
	dataFound.clear();
	bool ret = true;

	ret = Randomize();

	ret = CallKernel();

	// Get the result
	if (spinWait) {
		CudaSafeCall(hipMemcpy(outputBufferPinned, outputBuffer, outputSize, hipMemcpyDeviceToHost));
	}
	else {
		// Use hipMemcpyAsync to avoid default spin wait of hipMemcpy wich takes 100% CPU
		hipEvent_t evt;
		CudaSafeCall(hipEventCreate(&evt));
		CudaSafeCall(hipMemcpyAsync(outputBufferPinned, outputBuffer, 4, hipMemcpyDeviceToHost, 0));
		CudaSafeCall(hipEventRecord(evt, 0));
		while (hipEventQuery(evt) == hipErrorNotReady) {
			// Sleep 1 ms to free the CPU
			Timer::SleepMillis(1);
		}
		CudaSafeCall(hipEventDestroy(evt));
	}

	// Look for found
	uint32_t nbFound = outputBufferPinned[0];
	if (nbFound > maxFound) {
		nbFound = maxFound;
	}

	// When can perform a standard copy, the kernel is eneded
	CudaSafeCall(hipMemcpy(outputBufferPinned, outputBuffer, nbFound * ITEM_SIZE_A + 4, hipMemcpyDeviceToHost));

	for (uint32_t i = 0; i < nbFound; i++) {
		uint32_t* itemPtr = outputBufferPinned + (i * ITEM_SIZE_A32 + 1);
		ITEM it;
		it.thId = itemPtr[0];
		it.pubKey = (uint8_t*)(itemPtr + 1);
		it.hash160 = (uint8_t*)(itemPtr + 10);
		dataFound.push_back(it);
	}

	return ret;
}

// ----------------------------------------------------------------------------

bool GPUEngine::Randomize()
{
	// Properly use the range information for key generation
	if (use_range_) {
		// Initialize cuRAND states if not already initialized
		if (dev_rand_states_ == nullptr) {
			printf("Initializing cuRAND states for range-based search...\n");
			
			// Allocate memory for cuRAND states, one per thread
			CudaSafeCall(hipMalloc((void**)&dev_rand_states_, nbThread * sizeof(hiprandStatePhilox4_32_10_t)));
			
			// Initialize cuRAND states with current time as seed
			unsigned long long seed = (unsigned long long)std::time(0);
			int threadsPerBlock = 256;
			int blocks = (nbThread + threadsPerBlock - 1) / threadsPerBlock;
			
			init_curand_states_kernel<<<blocks, threadsPerBlock>>>(
				dev_rand_states_, seed, nbThread);
			
			CudaSafeCall(hipDeviceSynchronize());
			CudaSafeCall(hipGetLastError());
		}
		
		// Use the generate_keys_in_range_kernel to generate keys within the specified range
		int threadsPerBlock = 256;
		int blocks = (nbThread + threadsPerBlock - 1) / threadsPerBlock;
		
		generate_keys_in_range_kernel<<<blocks, threadsPerBlock>>>(
			inputKey, dev_rand_states_, dev_start_key_, dev_range_span_, nbThread);
		
		CudaSafeCall(hipDeviceSynchronize());
		CudaSafeCall(hipGetLastError());
		
		return true;
	} 
	else {
		// Fall back to the original simple randomization when no range is specified
		CudaSafeCall(hipMemset(inputKey, 0, nbThread * 4 * sizeof(uint64_t)));
		
		uint64_t* hostKeys = new uint64_t[nbThread * 4];
		for (int i = 0; i < nbThread * 4; i++) {
			uint64_t seed = std::time(0) ^ (i << 8);
			hostKeys[i] = seed;
		}
		
		CudaSafeCall(hipMemcpy(inputKey, hostKeys, nbThread * 4 * sizeof(uint64_t), hipMemcpyHostToDevice));
		delete[] hostKeys;
		
		return true;
	}
}

// ----------------------------------------------------------------------------

// Helper function to convert hex char to int
__host__ int hex_char_to_int(char c) {
	if (c >= '0' && c <= '9') return c - '0';
	if (c >= 'a' && c <= 'f') return c - 'a' + 10;
	if (c >= 'A' && c <= 'F') return c - 'A' + 10;
	return -1;
}

// Helper function: Host-side 256-bit hex string to uint64_t[4]
__host__ bool HostBN_HexToU64Array(const std::string& hex, uint64_t arr[4]) {
	if (hex.length() != 64) return false; // 256 bits = 32 bytes = 64 hex chars
	for (int i = 0; i < 4; ++i) arr[i] = 0;
	for (int i = 0; i < 4; ++i) { // Each u64 is 16 hex chars
		for (int j = 0; j < 16; ++j) {
			char c = hex[i * 16 + j];
			int val = hex_char_to_int(c);
			if (val == -1) return false;
			arr[3-i] |= (uint64_t)val << ((15 - j) * 4); // Fill from MSB u64 down, MSB char first
		}
	}
	// The above loop fills arr[3] with MSB ... arr[0] with LSB of the 256-bit number
	// Let's adjust to standard little-endian for u64 words (arr[0]=LSW, arr[3]=MSW)
	uint64_t temp_arr[4];
	for (int i = 0; i < 4; ++i) temp_arr[i] = 0;
	int hex_idx = 63;
	for(int arr_idx = 0; arr_idx < 4; ++arr_idx) { // For each uint64_t in the array (LSW to MSW)
		for(int char_idx = 0; char_idx < 16; ++char_idx) { // For each hex char in the uint64_t
			if(hex_idx < 0) break;
			int val = hex_char_to_int(hex[hex_idx--]);
			if(val == -1) return false;
			temp_arr[arr_idx] |= (uint64_t)val << (char_idx * 4);
		}
	}
	memcpy(arr, temp_arr, 4 * sizeof(uint64_t));
	return true;
}

// Host-side 256-bit subtraction: r = a - b. Returns borrow.
__host__ uint64_t HostBN_Sub(uint64_t r[4], const uint64_t a[4], const uint64_t b[4]) {
	uint64_t borrow = 0;
	for (int i = 0; i < 4; ++i) {
		uint64_t temp = a[i] - borrow;
		borrow = (a[i] < borrow); // Borrow from previous subtraction
		borrow += (temp < b[i]);  // Borrow for current subtraction
		r[i] = temp - b[i];
	}
	return borrow; // 1 if a < b, 0 otherwise
}

// Host-side 256-bit addition of 1: r = r + 1. Returns carry.
__host__ uint64_t HostBN_AddOneInplace(uint64_t r[4]) {
	uint64_t carry = 1;
	for (int i = 0; i < 4 && carry; ++i) {
		uint64_t old_val = r[i];
		r[i] += carry;
		if (r[i] < old_val) carry = 1; // overflow
		else carry = 0;
	}
	return carry;
}

// Device kernel to initialize cuRAND states
__global__ void init_curand_states_kernel(hiprandStatePhilox4_32_10_t *states, unsigned long long seed, int num_states) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < num_states) {
		hiprand_init(seed, tid, 0, &states[tid]);
	}
}

// Device function for 256-bit random number (fills r with 4 uint64_t)
// Uses Philox, which is good for parallel PRNG.
__device__ void DeviceBN_GetRandom256(hiprandStatePhilox4_32_10_t *state, uint64_t r[4]) {
	// Use hiprand to generate random 32-bit values and combine them into 64-bit
	uint4 r1, r2;
	// Use curand4 to generate 4 random 32-bit values at once
	r1.x = hiprand(state);
	r1.y = hiprand(state);
	r1.z = hiprand(state);
	r1.w = hiprand(state);
	r2.x = hiprand(state);
	r2.y = hiprand(state);
	r2.z = hiprand(state);
	r2.w = hiprand(state);
	
	// Combine 32-bit values into 64-bit values
	r[0] = ((uint64_t)r1.y << 32) | r1.x;
	r[1] = ((uint64_t)r1.w << 32) | r1.z;
	r[2] = ((uint64_t)r2.y << 32) | r2.x;
	r[3] = ((uint64_t)r2.w << 32) | r2.z;
}

// Device function for 256-bit addition: r = a + b (uint64_t[4])
// Uses PTX assembly macros for efficient carry handling.
__device__ void DeviceBN_Add256(uint64_t r[4], const uint64_t a[4], const uint64_t b[4]) {
	asm volatile ("add.cc.u64 %0, %1, %2;" : "=l"(r[0]) : "l"(a[0]), "l"(b[0]));
	asm volatile ("addc.cc.u64 %0, %1, %2;" : "=l"(r[1]) : "l"(a[1]), "l"(b[1]));
	asm volatile ("addc.cc.u64 %0, %1, %2;" : "=l"(r[2]) : "l"(a[2]), "l"(b[2]));
	asm volatile ("addc.u64 %0, %1, %2;" : "=l"(r[3]) : "l"(a[3]), "l"(b[3]));
}

// Device function for 256-bit subtraction: r = a - b. Returns borrow out of MSB.
// r = a - b. Returns 1 if a < b (borrow needed from MSB), 0 otherwise.
__device__ uint64_t DeviceBN_Sub256(uint64_t r[4], const uint64_t a[4], const uint64_t b[4]) {
	uint64_t borrow_out;
	asm volatile ("sub.cc.u64 %0, %1, %2;" : "=l"(r[0]) : "l"(a[0]), "l"(b[0]));
	asm volatile ("subc.cc.u64 %0, %1, %2;" : "=l"(r[1]) : "l"(a[1]), "l"(b[1]));
	asm volatile ("subc.cc.u64 %0, %1, %2;" : "=l"(r[2]) : "l"(a[2]), "l"(b[2]));
	asm volatile ("subc.u64 %0, %1, %2;" : "=l"(r[3]) : "l"(a[3]), "l"(b[3]));
	// Check final carry/borrow flag. This is tricky with only PTX subc.
	// A common way is to check if r > a after subtraction when b is non-zero.
	// Or, more directly, the carry flag from the last subc.u64 can be captured.
	// For simplicity, let's assume a full comparison for borrow check after subtraction, or rely on host side pre-check.
	// A true borrow out for a > b would mean r[3] (MSB) indicates this. 
	// If a < b, then the result will wrap around. 
	// The host side HostBN_Sub already checks for a < b via its return.
	// On device, we might need to check if any intermediate a[i]-borrow < b[i]. 
	// A simpler check: if (a < b) then (a-b) will be (MAX_UINT256 - (b-a) + 1)
	// For rejection sampling, we mainly need comparison, not the result of (R - M_span)
	// So, this Sub256 might not be directly used by rejection sampling, but comparison will.
	// Let's leave this as a standard subtract for now.
	// To get the borrow out (a < b?): compare a with b directly.
	// This function is just a - b, the borrow logic is separate for comparison.
	return 0; // Placeholder for borrow out, proper check is involved.
}

// Device function for 256-bit comparison: returns true if a >= b, false otherwise.
__device__ bool DeviceBN_IsGreaterOrEqual256(const uint64_t a[4], const uint64_t b[4]) {
	// Compare from Most Significant Word to Least Significant Word
	if (a[3] > b[3]) return true;
	if (a[3] < b[3]) return false;
	// a[3] == b[3]
	if (a[2] > b[2]) return true;
	if (a[2] < b[2]) return false;
	// a[2] == b[2]
	if (a[1] > b[1]) return true;
	if (a[1] < b[1]) return false;
	// a[1] == b[1]
	return (a[0] >= b[0]);
}

// Kernel to generate keys in a specified range using rejection sampling
__global__ void generate_keys_in_range_kernel(
	uint64_t* output_keys, 
	hiprandStatePhilox4_32_10_t* states,
	const uint64_t* dev_start_key,  
	const uint64_t* dev_range_span, 
	int num_keys_to_generate
) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= num_keys_to_generate) return;

	uint64_t random_val_256bit[4];
	uint64_t final_key_256bit[4];

	// Loop for rejection sampling
	// We need to ensure dev_range_span is not zero if it's used as an upper bound for random_val.
	// The span is (end - start + 1), so it should be at least 1.
	// If span is 0 (e.g. start > end, which should be caught by host), this loop is problematic.
	bool span_is_zero = true;
	for(int i=0; i<4; ++i) if(dev_range_span[i] != 0) {span_is_zero = false; break;}
	if(span_is_zero) { // Should not happen if host logic is correct (range_span >= 1)
		// Default to start_key or handle error
		for(int i=0; i<4; ++i) final_key_256bit[i] = dev_start_key[i];
	} else {
		do {
			DeviceBN_GetRandom256(&states[tid], random_val_256bit);
			// Generate R in [0, 2^256 - 1]. We want R in [0, dev_range_span - 1].
			// If random_val_256bit >= dev_range_span, regenerate.
		} while (DeviceBN_IsGreaterOrEqual256(random_val_256bit, dev_range_span));
		// Now, random_val_256bit is in the range [0, dev_range_span - 1]
		DeviceBN_Add256(final_key_256bit, dev_start_key, random_val_256bit);
	}

	uint64_t* key_ptr = output_keys + (tid * 4);
	key_ptr[0] = final_key_256bit[0];
	key_ptr[1] = final_key_256bit[1];
	key_ptr[2] = final_key_256bit[2];
	key_ptr[3] = final_key_256bit[3];
}

// ----------------------------------------------------------------------------

